#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
#include <omp.h>
#include <math.h>
#include <time.h>
#include "calculus.h"
#include "gpu_calculus.h"
#include "omp_calculus.h"
#include "seq_calculus.h"
#include "load_balance.h"


// imprime os resultados na tela
void print_results(long long k, long long M, double result_sum, double result_sub) {
    double result = 0;
    if (llabs(k) <= llabs(M)) result = (M >= 0) ? 1 : -1;
    printf("Erro no calculo com a soma: %lf\n", fabs(result - result_sum));
    printf("Erro no calculo com a subtracao: %lf\n", fabs(result - result_sub));
    printf("--------------------------------------------------------------------------------\n");
}


// executa a integracao de Monte Carlo usando CUDA + OpenMP
void perform_monte_carlo_hibrid(long long N, long long k, long long M, long long gpu_N, long long omp_N, int world_rank) {
    double f, f2, gpu_f, gpu_f2, omp_f, omp_f2, result_sum, result_sub, start, finish, elapsed;
    if (world_rank == 0) {
        start = omp_get_wtime();
        MPI_Send(&omp_N, 1, MPI_DOUBLE, 1, 0, MPI_COMM_WORLD);
        gpu_get_f(gpu_N, k, M, &gpu_f, &gpu_f2);
        MPI_Recv(&omp_f, 1, MPI_DOUBLE, 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        MPI_Recv(&omp_f2, 1, MPI_DOUBLE, 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

        f = (gpu_f*gpu_N + omp_f*omp_N) / N;
        f2 = (gpu_f2*gpu_N + omp_f2*omp_N) / N;

        result_sum = monte_carlo_sum(f, f2, N);
        result_sub = monte_carlo_sub(f, f2, N);

        finish = omp_get_wtime();
        elapsed = finish - start;
        printf("Tempo com balanceamento de carga em segundos: %lf\n", elapsed);
        print_results(k, M, result_sum, result_sub);

    } else if (world_rank == 1) {
        MPI_Recv(&omp_N, 1, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        omp_get_f(omp_N, k, M, &omp_f, &omp_f2);
        MPI_Send(&omp_f, 1, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD);
        MPI_Send(&omp_f2, 1, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD);
    }
}


// executa a integracao de Monte Carlo com CUDA
void perform_monte_carlo_gpu(long long N, long long k, long long M, int world_rank) {
    if (world_rank == 0) {
        double result_sum, result_sub, elapsed;
        elapsed = gpu_monte_carlo(N, k, M, &result_sum, &result_sub);
        printf("Tempo na GPU com uma thread na CPU em segundos: %lf\n", elapsed);
        print_results(k, M, result_sum, result_sub);
    }
}


// executa a integracao de Monte Carlo com OpenMP
void perform_monte_carlo_omp(long long N, long long k, long long M, int world_rank) {
    if (world_rank == 0) {
        double result_sum, result_sub, elapsed;
        elapsed = omp_monte_carlo(N, k, M, &result_sum, &result_sub);
        printf("Tempo na CPU com %d threads em segundos: %lf\n", omp_get_max_threads(), elapsed);
        print_results(k, M, result_sum, result_sub);
    }
}


// executa a integracao de Monte Carlo Sequencial
void perform_monte_carlo_seq(long long N, long long k, long long M, int world_rank) {
    if (world_rank == 0) {
        double result_sum, result_sub, elapsed;
        elapsed = seq_monte_carlo(N, k, M, &result_sum, &result_sub);
        printf("Tempo sequencial em segundos: %lf\n", elapsed);
        print_results(k, M, result_sum, result_sub);
    }
}


int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Use: %s <N> <k> <M>\n", argv[0]);
        exit(3);
    }
    srand((unsigned int)time(NULL));
    long long N = atoll(argv[1]);
    long long k = atoll(argv[2]);
    long long M = atoll(argv[3]);
    long long gpu_N, omp_N;
    int world_rank, world_size;

    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    if (world_size < 2) {
        fprintf(stderr, "O numero de processos deve ser maior do que 1 para %s\n", argv[0]);
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    get_n_samples(N, k, M, &gpu_N, &omp_N, world_rank);
    perform_monte_carlo_hibrid(N, k, M, gpu_N, omp_N, world_rank);
    perform_monte_carlo_gpu(N, k, M, world_rank);
    perform_monte_carlo_omp(N, k, M, world_rank);
    perform_monte_carlo_seq(N, k, M, world_rank);

    MPI_Finalize();
    return EXIT_SUCCESS;
}
