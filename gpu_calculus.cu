#include "hip/hip_runtime.h"
#include "gpu_calculus.h"
#include "calculus.h"
#include "math.h"
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#define BLOCKDIM 32
#define MAXARRAYSIZE 130000000 // supoe GPU RAM < 1GB

hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess)
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    return result;
}


// realiza a reducao do vetor g_idata e devolve o resultado em g_odata
__global__ void reduction(double *g_idata, long long n, double *g_odata) {
    extern __shared__ double sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        sdata[tid] = g_idata[i];

        __syncthreads();
        // do reduction in shared mem
        unsigned int s = blockDim.x / 2;
        while (s > 0) {
            if (tid < s)
                sdata[tid] += sdata[tid + s];
            __syncthreads();
            s /= 2;
        }

        // write result for this block to global mem
        if (tid == 0) g_odata[blockIdx.x] = sdata[0];
    }
}


// recebe um vetor x e calcula f(x) inplace
__global__ void calculate_fx(double *x, long long n, long long k, long long M) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) x[i] = (sin((2 * M + 1) * M_PI * x[i]) * cos(2 * M_PI * k * x[i])) / sin(M_PI * x[i]);
}


// recebe um vetor f(x) e calcula f(x)^2 inplace
__global__ void calculate_fx_2(double *fx, long long n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) fx[i] = fx[i] * fx[i];
}


// cria um vetor de tamanho n com x aleatório entre (0, 0.5]
static double *create_random_x(long long n) {
    double *x = create_empty_array(n);
    for (int i = 0; i < n; i++)
        x[i] = get_random_x();
    return x;
}


// recebe N, k, M e calcula <f> e <f^2>
void gpu_get_f(long long N, long long k, long long M, double *f, double *f2) {

    *f = *f2 = 0;

    for (long long task = N; task > 0; task -= MAXARRAYSIZE) {

        long long n = (task < MAXARRAYSIZE) ? task : MAXARRAYSIZE;
        long long grid_dim = (n + BLOCKDIM-1) / BLOCKDIM;
        double *x_h = create_random_x(n);
        double *result_h = (double *) malloc(grid_dim * sizeof(double));

        double *x_d, *result_d;
        checkCuda( hipMalloc((void **) &x_d, n * sizeof(double)) );
        checkCuda( hipMalloc((void **) &result_d, grid_dim * sizeof(double)) );

        // calcula f(x) no device
        checkCuda( hipMemcpy( x_d, x_h, n * sizeof(double), hipMemcpyHostToDevice) );
        calculate_fx<<<grid_dim, BLOCKDIM>>>(x_d, n, k, M);

        // reduz f(x) no device e termina no host
        reduction <<<grid_dim, BLOCKDIM, BLOCKDIM>>>(x_d, n, result_d);
        checkCuda( hipMemcpy( result_h, result_d, grid_dim * sizeof(double), hipMemcpyDeviceToHost) );;
        for (int i = 0; i < grid_dim; i++)
            *f += result_h[i];

        // calcula f(x)^2 no device
        calculate_fx_2<<<grid_dim, BLOCKDIM>>>(x_d, n);

        // reduz f(x)^2 no device e termina no host
        reduction <<<grid_dim, BLOCKDIM, BLOCKDIM>>>(x_d, n, result_d);
        checkCuda( hipMemcpy( result_h, result_d, grid_dim * sizeof(double), hipMemcpyDeviceToHost) );
        for (int i = 0; i < grid_dim; i++)
            *f2 += result_h[i];

        // limpeza
        checkCuda( hipFree(x_d) );
        checkCuda( hipFree(result_d) );
        free(x_h);
        free(result_h);
    }

    *f /= N;  // encontra <f>
    *f2 /= N; // encontra <f^2>
}


// recebe N, k, M, calcula os dois resultados da integral de Monte Carlo e devolve o tempo de execucao
double gpu_monte_carlo(long long N, long long k, long long M, double *result_sum, double *result_sub) {
    double f, f2, start, finish;
    start = omp_get_wtime();
    gpu_get_f(N, k, M, &f, &f2);
    *result_sum = monte_carlo_sum(f, f2, N);
    *result_sub = monte_carlo_sub(f, f2, N);
    finish = omp_get_wtime();
    return finish - start;
}
