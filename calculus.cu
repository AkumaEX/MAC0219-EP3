#include "hip/hip_runtime.h"
#include "calculus.h"
#include <math.h>
#include <stdlib.h>

double get_random_x() {
    return ((double)(rand() + 1)) / 2.0 / ((double)RAND_MAX) ;
}

double monte_carlo_sum(double f, double f_2, long long N) {
    return f + sqrt((f_2 - (f * f)) / N );
}

double monte_carlo_sub(double f, double f_2, long long N) {
    return f - sqrt((f_2 - (f * f)) / N );
}

double *create_empty_array(long long N) {
    return (double *) calloc(N, sizeof(double));
}
