#include "hip/hip_runtime.h"
#include "omp_calculus.h"
#include "calculus.h"
#include <math.h>
#include <omp.h>
#include <stdlib.h>
#define M_PI 3.14159265358979323846
#define MAXARRAYSIZE 260000000 // supoe RAM principal < 2GB


// realiza a reducao do vetor idata
static double reduction(double *idata, long long n) {
    double sum = 0;
#pragma omp parallel for reduction (+:sum)
    for (int i = 0; i < n; i++)
        sum += idata[i];
    return sum;
}


// recebe um vetor de x de tamanho n e devolve f(x) inplace
static double *calculate_fx(double *x, long long n, long long k, long long M) {
#pragma omp parallel for
    for (int i = 0; i < n; i++)
        x[i] = (sin((2 * M + 1) * M_PI * x[i]) * cos(2 * M_PI * k * x[i])) / sin(M_PI * x[i]);
    return x;
}


// recebe um vetor de f(x) de tamanho n e devolve f(x)^2 inplace
static double *calculate_fx_2(double *fx, long long n) {
#pragma omp parallel for
    for (int i = 0; i < n; i++)
        fx[i] = fx[i] * fx[i];
    return fx;
}


// cria um vetor x aleatorio de tamanho n
static double *create_random_x(long long n) {
    double *x = create_empty_array(n);
#pragma omp parallel for
    for (int i = 0; i < n; i++)
        x[i] = get_random_x();
    return x;
}


// recebe N, k, M e realiza o calculo de <f> e <f^2>
void omp_get_f(long long N, long long k, long long M, double *f, double *f2) {

    *f = *f2 = 0;

    for (long long task = N; task > 0; task -= MAXARRAYSIZE) {
        long long n = (task < MAXARRAYSIZE) ? task : MAXARRAYSIZE;
        double *x = create_random_x(n);
        double *fx = calculate_fx(x, n, k, M);
        *f += reduction(fx, n);
        double *fx_2 = calculate_fx_2(fx, n);
        *f2 += reduction(fx_2, n);
        free(x);
    }

    *f /= N;
    *f2 /= N;
}


// recebe N, k, M, calcula os dois resultados da integral de Monte Carlo e devolve o tempo de execucao
double omp_monte_carlo(long long N, long long k, long long M, double *result_sum, double *result_sub) {
    double f, f2, start, finish;
    start = omp_get_wtime();
    omp_get_f(N, k, M, &f, &f2);
    *result_sum = monte_carlo_sum(f, f2, N);
    *result_sub = monte_carlo_sub(f, f2, N);
    finish = omp_get_wtime();
    return finish - start;
}