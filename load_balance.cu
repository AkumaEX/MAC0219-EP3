#include "hip/hip_runtime.h"
#include "load_balance.h"
#include "gpu_calculus.h"
#include "omp_calculus.h"
#include "seq_calculus.h"
#include <stdio.h>
#include <mpi.h>
#include <omp.h>

#define N_TRAIN 500


// devolve a taxa de crescimento da funcao linear dados dois pontos
double get_ratio(double y, double y0, long long x, long long x0) {
    return (y - y0) / (x - x0);
}


// recebe dados de tempo de execucao e devolve a quantidade de amostras que a GPU devera executar dado N
long long predict_gpu_n(double gpu_y0, double gpu_y, double omp_y0, double omp_y, long long x0, long long x, long long N) {
    double gpu_ratio = get_ratio(gpu_y, gpu_y0, x, x0);
    double omp_ratio = get_ratio(omp_y, omp_y0, x, x0);
    return (long long)(N * omp_ratio / (gpu_ratio + omp_ratio));
}


// calcula o numero de amostras para GPU (gpu_N) e OMP (omp_N)
void get_n_samples(long long N, long long k, long long M, long long *gpu_N, long long *omp_N, int world_rank) {
    *gpu_N = *omp_N = 0;
    long long x0 = 1;
    long long x = 100000;
    double gpu_y0, gpu_y, omp_y0, omp_y, result_sum, result_sub;
    gpu_y0 = gpu_y = omp_y0 = omp_y = 0.0;

    if (world_rank == 0) {
        printf("treinando ... \n");
        for (int i = 0; i < N_TRAIN; i++) {
            gpu_y0 += gpu_monte_carlo(x0, k, M, &result_sum, &result_sub)  / N_TRAIN;
            gpu_y  += gpu_monte_carlo( x, k, M, &result_sum, &result_sub)  / N_TRAIN;
        }
        MPI_Recv(&omp_y0, 1, MPI_DOUBLE, 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        MPI_Recv(&omp_y, 1, MPI_DOUBLE, 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

        *gpu_N = predict_gpu_n(gpu_y0, gpu_y, omp_y0, omp_y, x0, x, N);
        *omp_N = N - *gpu_N;
        printf("gpu_N: %lld, omp_N: %lld\n", *gpu_N, *omp_N);
        printf("--------------------------------------------------------------------------------\n");

    } else if (world_rank == 1) {
        for (int i = 0; i < N_TRAIN; i++) {
            omp_y0 += omp_monte_carlo(x0, k, M, &result_sum, &result_sub) / N_TRAIN;;
            omp_y  += omp_monte_carlo( x, k, M, &result_sum, &result_sub) / N_TRAIN;;
        }
        MPI_Send(&omp_y0, 1, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD);
        MPI_Send(&omp_y , 1, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD);
    }
}
